#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <assert.h>
#include "GLCM.h"
#include "GrayPair.h"
#include "AggregatedGrayPair.h"

using namespace std;

// Constructors
__device__ GLCM::GLCM(const unsigned int * pixels, const ImageData& image,
        Window& windowData, WorkArea& wa): pixels(pixels), img(image),
        windowData(windowData),  workArea(wa) ,grayPairs(wa.grayPairs),
        summedPairs(wa.summedPairs), subtractedPairs(wa.subtractedPairs),
        xMarginalPairs(wa.xMarginalPairs), yMarginalPairs(wa.yMarginalPairs)
        {
    // Computing the number of pairs that need to be processed in this GLCM
    this->numberOfPairs = getWindowRowsBorder() * getWindowColsBorder();

    // Dealing with memory leaking
    workArea.cleanup();
    // Generating the elements of this GLCM
    initializeGlcmElements();}


__device__ GLCM::~GLCM(){

}

__device__ int GLCM::getNumberOfPairs() const {
        return numberOfPairs;
}

__device__ int GLCM::getMaxGrayLevel() const {
    return img.getMaxGrayLevel();
}

/**
 * Geometric limit of the sub-window
 * @return the number of rows of the window to be considered
 */
__device__ int GLCM::getWindowRowsBorder() const{
   return (windowData.side - (windowData.distance * abs(windowData.shiftRows)));
}

/**
 * Geometric limit of the sub-window
 * @return the number of the columns of the window to be considered
 */
__device__ int GLCM::getWindowColsBorder() const{
    return (windowData.side - (windowData.distance * abs(windowData.shiftColumns)));
}


/**
 * Computing the shift to apply at the column for locating the pixels of each
 * pair of the glcm; it affects only 135° orientation
 * @return d (distance) pixels to be ignored
 */
__device__ inline int GLCM::computeWindowColumnOffset()
{
    int initialColumnOffset = 0; // for 0°,45°,90°
    if((windowData.shiftRows * windowData.shiftColumns) > 0) // 135°
        initialColumnOffset = 1;
    return initialColumnOffset;
}

/**
 * Computes the shift to apply at the row for locating the pixels of each
 * pair of the glcm; it does not affect 0° orientation alone
 * @return d (distance) pixels need to be ignored
*/
__device__ inline int GLCM::computeWindowRowOffset()
{
    int initialRowOffset = 1; // for 45°,90°,135°
    if((windowData.shiftRows == 0) && (windowData.shiftColumns > 0))
        initialRowOffset = 0; // for 0°
    return initialRowOffset;
}

/**
 * Methods to obtain the reference pixel in each pair of the glcm
 * @param row in the sub-window of the reference pixel
 * @param col in the sub-window of the reference pixel
 * @param initialRowOffset see computeWindowRowOffset
 * @param initialColumnOffset see computeWindowColOffset
 * @return the index of the pixel in the array of pixels (linearized) of
 * the window
 */
__device__ inline int GLCM::getReferenceIndex(const int i, const int j,
                                   const int initialWindowRowOffset, const int initialWindowColumnOffset){
    int row = (i + windowData.imageRowsOffset) // starting point in the image
            + (initialWindowRowOffset * windowData.distance); // add direction eventual down-shift (45°, 90°, 135°)
    int col = (j + windowData.imageColumnsOffset) + // starting point in the image
            (initialWindowColumnOffset * windowData.distance); // add direction shift
    int index = ( row * img.getColumns()) + col;
    assert(index >= 0);
    return index;
}

/**
 * Methods to obtain the neighbor pixel in each pair of the glcm
 * @param row in the sub-window of the neighbor pixel
 * @param col in the sub-window of the neighbor pixel
 * @param initialColumnOffset see computeWindowColOffset
 * @return the index of the pixel in the array of pixels (linearized) of
 * the window
 */
__device__ inline int GLCM::getNeighborIndex(const int i, const int j,
                                  const int initialWindowColumnOffset){
    int row = (i + windowData.imageRowsOffset); // starting point in the image
    int col = (j + windowData.imageColumnsOffset) + // starting point in the image
              (initialWindowColumnOffset * windowData.distance) +  // add 135* right-shift
              (windowData.shiftColumns * windowData.distance); // add direction shift
    int index = (row * img.getColumns()) + col;
    assert(index >= 0);
    return index;
}

/**
 * Method that adds a GrayPair into the pre-allocated memory
 * It uses the convention that GrayPair (i=0, j=0, frequency=0) means
 * available memory
 */
__device__ inline void GLCM::insertElement(GrayPair* grayPairs, const GrayPair actualPair, 
    uint32_t& lastInsertionPosition, bool symmetry){
    int position = 0;
    // Finding if the element was already inserted, and where
    while((!grayPairs[position].compareTo(actualPair, symmetry)) && (position < numberOfPairs))
        position++;
    // If found
    if((lastInsertionPosition > 0) // 0,0 as first element will increase insertion position
        && (position != numberOfPairs)){ // if the item was already inserted
        grayPairs[position].operator++();
        if((actualPair.getGrayLevelI() == 0) && (actualPair.getGrayLevelJ() == 0)
            && (grayPairs[position].getFrequency() == actualPair.getFrequency()))
            // Corner case, the inserted pair <0,0> that matches with every empty field
            lastInsertionPosition++;
    }
    else
    {
        grayPairs[lastInsertionPosition] = actualPair;
        lastInsertionPosition++;
    }
}

/**
 * This method creates the array of GrayPairs
*/
__device__ void GLCM::initializeGlcmElements() {
    // Defining the subBorders offset according to the orientation
    int initialWindowColumnOffset = computeWindowColumnOffset();
    int initialWindowRowOffset = computeWindowRowOffset();

    grayLevelType referenceGrayLevel;
    grayLevelType neighborGrayLevel;
    unsigned int lastInsertionPosition = 0;
    // Navigating the sub-window of interest
    for (int i = 0; i < getWindowRowsBorder() ; i++)
    {
        for (int j = 0; j < getWindowColsBorder(); j++)
        {
            // Extracting the two pixels in the pair
            int referenceIndex = getReferenceIndex(i, j,
                    initialWindowRowOffset, initialWindowColumnOffset);
            // Limit up to 2^16 gray levels
            referenceGrayLevel = pixels[referenceIndex]; // should be safe
            int neighborIndex = getNeighborIndex(i, j,
                    initialWindowColumnOffset);
            // Limit up to 2^16 gray levels
            neighborGrayLevel = pixels[neighborIndex];  // should be safe

            GrayPair actualPair;
            
            if((windowData.symmetric) && (neighborGrayLevel > referenceGrayLevel))
            {
                actualPair = GrayPair(neighborGrayLevel, referenceGrayLevel);
            }
            else
            {
                actualPair = GrayPair(referenceGrayLevel, neighborGrayLevel);
            }

            insertElement(grayPairs, actualPair, lastInsertionPosition, windowData.symmetric);
            
        }
    }
    effectiveNumberOfGrayPairs = lastInsertionPosition;
    codifyAggregatedPairs();
    codifyMarginalPairs();
}

/**
 * Method that adds an AggregatedGrayPair into the pre-allocated memory.
 * It uses the convention that AggregateGrayPair (k=0, frequency=0) means
 * available memory
 */
__device__ inline void GLCM::insertElement(AggregatedGrayPair* elements, const AggregatedGrayPair actualPair, uint32_t& lastInsertionPosition){
    int position = 0;
    // Finding if the element was already inserted, and where
    while((!elements[position].compareTo(actualPair)) && (position < numberOfPairs))
        position++;
    // If found
    if((lastInsertionPosition > 0) && // corner case 0 as first element
        (position != numberOfPairs)){ // if the item was already inserted
            elements[position].increaseFrequency(actualPair.getFrequency());
        if((actualPair.getAggregatedGrayLevel() == 0) && // corner case 0 as regular element
        (elements[position].getFrequency() == actualPair.getFrequency()))
            // Corner case, inserted 0 that matches with every empty field
            lastInsertionPosition++;
    }
    else
    {
        elements[lastInsertionPosition] = actualPair;
        lastInsertionPosition++;
    }
}

/**
 * This method produces the two arrays of AggregatedPairs (k, frequency)
 * where k is the sum or difference of both grayLevels of a GrayPair.
 * This representation is used in computeSumXXX() and computeDiffXXX() features
*/
__device__ void GLCM::codifyAggregatedPairs() {
    unsigned int lastInsertPosition = 0;
    // Summed pairs, firstly
    for(int i = 0 ; i < effectiveNumberOfGrayPairs; i++){
        // Creating the summed pairs, firstly
        grayLevelType k= grayPairs[i].getGrayLevelI() + grayPairs[i].getGrayLevelJ();
        AggregatedGrayPair summedElement(k, grayPairs[i].getFrequency());

        insertElement(summedPairs, summedElement, lastInsertPosition);
    }
    numberOfSummedPairs = lastInsertPosition;

    // Diff pairs, secondly
    lastInsertPosition = 0;
    for(int i = 0 ; i < effectiveNumberOfGrayPairs; i++){
        int diff = grayPairs[i].getGrayLevelI() - grayPairs[i].getGrayLevelJ();
        grayLevelType k= static_cast<uint32_t>(abs(diff));
        AggregatedGrayPair element(k, grayPairs[i].getFrequency());

        insertElement(subtractedPairs, element, lastInsertPosition);
    }
    numberOfSubtractedPairs = lastInsertPosition;
}

/**
 * This method produces the two arrays of AggregatedPairs (k, frequency)
 * where k is one grayLevel of GLCM and frequency is the "marginal" frequency of that level
 * (i.e., how many times k is present in all GrayPair<k, ?>)
 * This representation is used for computing features HX, HXY, HXY1, imoc
*/
__device__ void GLCM::codifyMarginalPairs() {
    unsigned int lastInsertPosition = 0;
    // xMarginalPairs, firstly
    for(int i = 0 ; i < effectiveNumberOfGrayPairs; i++){
        grayLevelType firstGrayLevel = grayPairs[i].getGrayLevelI();
        AggregatedGrayPair element(firstGrayLevel, grayPairs[i].getFrequency());

        insertElement(xMarginalPairs, element, lastInsertPosition);
    }
    numberOfxMarginalPairs = lastInsertPosition;

    // yMarginalPairs, secondly
    lastInsertPosition = 0;
    for(int i = 0 ; i < effectiveNumberOfGrayPairs; i++){
        grayLevelType secondGrayLevel = grayPairs[i].getGrayLevelJ();
        AggregatedGrayPair element(secondGrayLevel, grayPairs[i].getFrequency());

        insertElement(yMarginalPairs, element, lastInsertPosition);
    }
    numberOfyMarginalPairs = lastInsertPosition;
}

/* DEBUGGING METHODS */
__device__ void GLCM::printGLCM() const {
    printGLCMData();
    printGLCMElements();
    printAggregated();
    printMarginalProbabilityElements();
}

__device__ void GLCM::printGLCMData() const{
    printf("\n");
    printf("***\tGLCM data\t***\n");
    printf("Shift rows: %d \n", windowData.shiftRows);
    printf("Shift columns: %d \n", windowData.shiftColumns);
    printf("Sliding window side: %d \n", windowData.side);
    printf("Border rows: %d \n", getWindowRowsBorder());
    printf("Border columns: %d \n", getWindowColsBorder());
    printf("Symetric: ");
    if(windowData.symmetric){
    	printf("Yes\n");
    }
    else{
    	printf("No\n");
    }
    printf("\n");;
}

__device__ void GLCM::printGLCMElements() const{
    printf("* GrayPairs *\n");
    for (int i = 0; i < effectiveNumberOfGrayPairs; ++i) {
        grayPairs[i].printPair();;
    }
}

__device__ void GLCM::printAggregated() const{
    printGLCMAggregatedElements(true);
    printGLCMAggregatedElements(false);
}

__device__ void GLCM::printGLCMAggregatedElements(bool areSummed) const{
    printf("\n");
    if(areSummed) {
        printf("* Summed grayPairsMap *\n");
        for (int i = 0; i < numberOfSummedPairs; ++i) {
            summedPairs[i].printPair();
        }
    }
    else {
        printf("* Subtracted grayPairsMap *\n");
        for (int i = 0; i < numberOfSubtractedPairs; ++i) {
            subtractedPairs[i].printPair();
        }
    }
}



__device__ void GLCM::printMarginalProbabilityElements() const{
    printf("\n* xMarginal encoding\n");
    for (int i = 0; i < numberOfxMarginalPairs; ++i) {
        printf("(%d, X):\t%d\n", xMarginalPairs[i].getAggregatedGrayLevel(), xMarginalPairs[i].getFrequency());
    }
    printf("\n* yMarginal encoding\n");
    for (int i = 0; i <numberOfyMarginalPairs; ++i) {
        printf("(X, %d):\t%d\n", yMarginalPairs[i].getAggregatedGrayLevel(), yMarginalPairs[i].getFrequency());

    }

}


